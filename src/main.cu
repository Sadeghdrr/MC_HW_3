#include "hip/hip_runtime.h"
#ifndef _GNU_SOURCE
#define _GNU_SOURCE  // For getline and strdup
#endif
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <pthread.h>
#include <hs/hs.h>
#include <sys/stat.h>
#include <unistd.h>

// CUDA includes for GPU mode
#include <hip/hip_runtime.h>
#include <>

// cuDF RAPIDS includes for GPU regex
#include <rmm/device_uvector.hpp>
#include <rmm/cuda_stream_view.hpp>
#include <rmm/mr/device/per_device_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/strings/contains.hpp>
#include <cudf/strings/regex/regex_program.hpp>
#include <cudf/types.hpp>

// CUDA error checking macro with detailed error reporting
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Error detected. %s %s\n", hipGetErrorName(err), hipGetErrorString(err)); \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Additional macro for checking CUDA errors after kernel launches
#define CUDA_CHECK_KERNEL() \
    do { \
        hipError_t err = hipGetLastError(); \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA Kernel Error detected. %s %s\n", hipGetErrorName(err), hipGetErrorString(err)); \
            fprintf(stderr, "CUDA kernel error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)


// --- Data Structures ---

// Execution Mode
typedef enum {
    MODE_CPU,
    MODE_GPU
} execution_mode_t;

// Configuration Structure
typedef struct {
    execution_mode_t mode;
    char* rules_file;
    char* input_file;
    int num_threads;  // Only used for CPU mode
    int gpu_streams;  // GPU: number of concurrent pattern workers (logical)
    int gpu_wave;      // GPU: patterns per batch
    int gpu_chunk_mb;  // GPU: per-chunk input size in MB
} config_t;

/**
 * @struct MatchContext
 * @brief Context structure passed to the Hyperscan match event handler.
 */
typedef struct {
    int* matches;           // Array to store IDs of matched rules.
    int match_count;        // Number of matches found for the current line.
    int match_capacity;     // Allocated capacity of the matches array.
} MatchContext;

/**
 * @struct ThreadData
 * @brief Data structure to pass information to each worker thread.
 */
typedef struct {
    int thread_id;                 // Unique identifier for the thread.
    char** lines;                  // Pointer to the array of all input lines.
    unsigned int* line_lengths;    // Pointer to the array of all line lengths.
    long start_line;               // Starting line index for this thread.
    long end_line;                 // Ending line index for this thread.
    hs_database_t* database;       // Pointer to the compiled Hyperscan database.
    hs_scratch_t* scratch;         // Per-thread scratch space for Hyperscan.
    char*** thread_results;        // 2D array: [line_index][match_list] for this thread's lines
    long total_matches;            // Total number of matches found by this thread.
} ThreadData;


// --- Forward Declarations ---
int run_cpu_mode(const config_t* config);
int run_gpu_mode(const config_t* config);

// cuDF helper functions for GPU mode
#ifdef __cplusplus
extern "C" {
#endif

// Build device strings column from host vector<string>
static std::unique_ptr<cudf::column>
make_device_strings(const std::vector<std::string>& h, rmm::cuda_stream_view stream) {
    using size_type = cudf::size_type;
    const size_type n = static_cast<size_type>(h.size());

    // Handle edge case of empty input
    if (n == 0) {
        return cudf::make_empty_column(cudf::data_type{cudf::type_id::STRING});
    }

    std::vector<int32_t> h_offsets(n + 1, 0);
    size_t total_chars = 0;
    for (size_t i = 0; i < h.size(); ++i) {
        total_chars += h[i].size();
        h_offsets[i + 1] = static_cast<int32_t>(total_chars);
    }
    
    std::vector<char> h_chars;
    h_chars.reserve(total_chars);
    for (auto& s : h) h_chars.insert(h_chars.end(), s.begin(), s.end());

    // Allocate device memory with explicit error checking
    rmm::device_uvector<int32_t> d_offsets(n + 1, stream);
    rmm::device_uvector<char> d_chars(total_chars, stream);

    CUDA_CHECK(hipMemcpyAsync(d_offsets.data(), h_offsets.data(),
                               (n + 1) * sizeof(int32_t),
                               hipMemcpyHostToDevice, stream.value()));
    if (total_chars > 0) {
        CUDA_CHECK(hipMemcpyAsync(d_chars.data(), h_chars.data(), total_chars,
                                   hipMemcpyHostToDevice, stream.value()));
    }
    
    // Synchronize to ensure data transfer is complete
    CUDA_CHECK(hipStreamSynchronize(stream.value()));

    auto null_mask = rmm::device_buffer{0, stream};
    cudf::size_type null_count = 0;

    auto offsets_buf = d_offsets.release();
    auto offsets_col = std::make_unique<cudf::column>(
        cudf::data_type{cudf::type_id::INT32},
        n + 1,
        std::move(offsets_buf),
        rmm::device_buffer{0, stream},
        0);
    auto chars_buf = d_chars.release();
    return cudf::make_strings_column(
        n,
        std::move(offsets_col),
        std::move(chars_buf),
        null_count,
        std::move(null_mask));
}

__global__ void add_true_to_counts(const uint8_t* __restrict__ vals,
                                   int n,
                                   int* __restrict__ counts) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) counts[i] += (vals[i] != 0);
}

#ifdef __cplusplus
}
#endif

// --- Utility Functions ---

/**
 * @brief Prints an error message and exits the program.
 */
void fail(const char* msg) {
    fprintf(stderr, "ERROR: %s\n", msg);
    exit(EXIT_FAILURE);
}

/**
 * @brief Print usage information.
 */
void print_usage(const char* program_name) {
    printf("Usage: %s --mode <cpu|gpu> --rules <rules_file> --input <input_file> [--threads <n>] [--gpu-streams <n>] [--gpu-wave <n>] [--gpu-chunk-mb <n>]\n", program_name);
    printf("\nRequired arguments:\n");
    printf("  --mode      <cpu|gpu>       Processing mode (CPU or GPU)\n");
    printf("  --rules     <rules_file>    Path to the rules file\n");
    printf("  --input     <input_file>    Path to the input file\n");
    printf("\nOptional arguments:\n");
    printf("  --threads   <num_threads>   Number of threads (required for CPU mode)\n");
    printf("  --gpu-streams <n>        GPU: logical concurrency across patterns (default 6)\n");
    printf("  --gpu-wave    <n>        GPU: patterns per batch (default 48)\n");
    printf("  --gpu-chunk-mb <n>       GPU: max input MB per chunk (default 128)\n");
    printf("\nOutput files are automatically generated in the results/ directory:\n");
    printf("  Results_HW3_MCC_030402_401106039_{CPU/GPU}_{DataSet}_{NumThreads/Library}.txt\n");
    printf("  Results_HW3_MCC_030402_401106039_{CPU/GPU}_{DataSet}_{Hyperscan/GPULibrary}.csv\n");
    printf("\nExample:\n");
    printf("  %s --mode cpu --rules rules.txt --input set1.txt --threads 4\n", program_name);
    printf("  %s --mode gpu --rules rules.txt --input set1.txt --gpu-streams 6 --gpu-wave 48 --gpu-chunk-mb 128\n", program_name);
    exit(EXIT_FAILURE);
}

/**
 * @brief Generate automatic output filename based on configuration.
 */
char* generate_output_filename(const config_t* config) {
    // Extract dataset name from input file (e.g., "set1.txt" -> "set1")
    const char* input_basename = strrchr(config->input_file, '/');
    if (input_basename) {
        input_basename++; // Skip the '/'
    } else {
        input_basename = config->input_file;
    }
    
    // Remove file extension
    char dataset[256];
    strncpy(dataset, input_basename, sizeof(dataset) - 1);
    dataset[sizeof(dataset) - 1] = '\0';
    char* dot = strrchr(dataset, '.');
    if (dot) {
        *dot = '\0';
    }
    
    // Allocate memory for the filename
    char* filename = (char*)malloc(512);
    if (!filename) {
        fprintf(stderr, "Error: Memory allocation failed for output filename\n");
        exit(EXIT_FAILURE);
    }
    
    if (config->mode == MODE_CPU) {
        snprintf(filename, 512, "results/Results_HW3_MCC_030402_401106039_CPU_%s_%d.txt", 
                 dataset, config->num_threads);
    } else {
        snprintf(filename, 512, "results/Results_HW3_MCC_030402_401106039_GPU_%s_CUDA.txt", 
                 dataset);
    }
    
    return filename;
}

/**
 * @brief Generate performance CSV filename based on configuration.
 */
char* generate_performance_filename(const config_t* config, const char* input_filename) {
    // Extract dataset name from input filename
    const char* dataset_name = strrchr(input_filename, '/');
    if (dataset_name) {
        dataset_name++; // Skip the '/'
    } else {
        dataset_name = input_filename;
    }
    
    // Remove extension from dataset name
    char* dataset_clean = strdup(dataset_name);
    char* dot = strrchr(dataset_clean, '.');
    if (dot) *dot = '\0';
    
    char* filename = (char*)malloc(512);
    if (config->mode == MODE_CPU) {
        snprintf(filename, 512, "results/Results_HW3_MCC_030402_401106039_CPU_%s_Hyperscan.csv", 
                 dataset_clean);
    } else {
        snprintf(filename, 512, "results/Results_HW3_MCC_030402_401106039_GPU_%s_CUDA.csv", 
                 dataset_clean);
    }
    
    free(dataset_clean);
    return filename;
}

/**
 * @brief Parse command line arguments.
 */
config_t parse_arguments(int argc, char* argv[]) {
    config_t config{};
    // Defaults for GPU tuning
    config.gpu_streams = 6;
    config.gpu_wave    = 48;
    config.gpu_chunk_mb= 128;

    
    if (argc < 5) {  // Minimum required arguments for GPU mode
        print_usage(argv[0]);
    }
    
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--mode") == 0 && i + 1 < argc) {
            if (strcmp(argv[i + 1], "cpu") == 0) {
                config.mode = MODE_CPU;
            } else if (strcmp(argv[i + 1], "gpu") == 0) {
                config.mode = MODE_GPU;
            } else {
                fprintf(stderr, "ERROR: Invalid mode '%s'. Use 'cpu' or 'gpu'.\n", argv[i + 1]);
                print_usage(argv[0]);
            }
            i++; // Skip next argument
        } else if (strcmp(argv[i], "--rules") == 0 && i + 1 < argc) {
            config.rules_file = argv[i + 1];
            i++;
        } else if (strcmp(argv[i], "--input") == 0 && i + 1 < argc) {
            config.input_file = argv[i + 1];
            i++;
        } else if (strcmp(argv[i], "--threads") == 0 && i + 1 < argc) {
            config.num_threads = atoi(argv[i + 1]); i++;
        } else if (strcmp(argv[i], "--gpu-streams") == 0 && i + 1 < argc) {
            config.gpu_streams = atoi(argv[i + 1]); i++;
        } else if (strcmp(argv[i], "--gpu-wave") == 0 && i + 1 < argc) {
            config.gpu_wave = atoi(argv[i + 1]); i++;
        } else if (strcmp(argv[i], "--gpu-chunk-mb") == 0 && i + 1 < argc) {
            config.gpu_chunk_mb = atoi(argv[i + 1]); i++;
            config.num_threads = atoi(argv[i + 1]);
            if (config.num_threads <= 0) {
                fprintf(stderr, "ERROR: Number of threads must be a positive integer.\n");
                print_usage(argv[0]);
            }
            i++;
        }
    }
    
    // Validate required arguments
    if (!config.rules_file || !config.input_file) {
        fprintf(stderr, "ERROR: Missing required arguments.\n");
        print_usage(argv[0]);
    }
    
    if (config.mode == MODE_CPU && config.num_threads == 0) {
        fprintf(stderr, "ERROR: --threads argument is required for CPU mode.\n");
        print_usage(argv[0]);
    }
    
    return config;
}

/**
 * @brief Reads all lines from a file into a dynamically allocated array.
 */
char** read_lines_from_file(const char* filename, long* line_count, unsigned int** line_lengths, long* total_bytes) {
    FILE* file = fopen(filename, "r");
    if (!file) {
        perror("fopen failed");
        fail("Could not open file.");
    }

    // Get file size for total_bytes metric
    struct stat st;
    if (stat(filename, &st) == 0) {
        *total_bytes = st.st_size;
    } else {
        *total_bytes = 0; // Fallback
    }

    long capacity = 1024;
    char** lines = (char**)malloc(capacity * sizeof(char*));
    if (!lines) fail("Failed to allocate memory for lines.");

    *line_count = 0;
    char* line_buffer = NULL;
    size_t buffer_size = 0;

    while (getline(&line_buffer, &buffer_size, file) != -1) {
        if (*line_count >= capacity) {
            capacity *= 2;
            lines = (char**)realloc(lines, capacity * sizeof(char*));
            if (!lines) fail("Failed to reallocate memory for lines.");
        }
        // Strip newline characters
        line_buffer[strcspn(line_buffer, "\r\n")] = 0;
        lines[*line_count] = strdup(line_buffer);
        if (!lines[*line_count]) fail("Failed to duplicate line.");
        (*line_count)++;
    }

    free(line_buffer);
    fclose(file);

    // Create the line lengths array
    *line_lengths = (unsigned int*)malloc(*line_count * sizeof(unsigned int));
    if (!*line_lengths) fail("Failed to allocate memory for line lengths.");
    for (long i = 0; i < *line_count; i++) {
        (*line_lengths)[i] = strlen(lines[i]);
    }

    return lines;
}

// --- Hyperscan Match Callback ---

/**
 * @brief Hyperscan match event handler.
 */
static int onMatch(unsigned int id, unsigned long long from, unsigned long long to,
                   unsigned int flags, void* ctx) {
    (void)from;   // Suppress unused parameter warning
    (void)to;     // Suppress unused parameter warning
    (void)flags;  // Suppress unused parameter warning
    
    MatchContext* context = (MatchContext*)ctx;

    // Resize matches array if needed
    if (context->match_count >= context->match_capacity) {
        context->match_capacity *= 2;
        context->matches = (int*)realloc(context->matches, context->match_capacity * sizeof(int));
        if (!context->matches) {
            fail("Failed to reallocate memory for matches in callback.");
        }
    }

    context->matches[context->match_count++] = id;
    return 0; // Continue scanning
}


// --- Worker Thread ---

/**
 * @brief The main function for each worker thread.
 */
void* worker_thread(void* arg) {
    ThreadData* data = (ThreadData*)arg;
    data->total_matches = 0;

    // Allocate scratch space for this thread
    hs_error_t scratch_err = hs_alloc_scratch(data->database, &data->scratch);
    if (scratch_err != HS_SUCCESS) {
        fprintf(stderr, "Thread %d: Failed to allocate scratch space. Error: %d\n", data->thread_id, scratch_err);
        return NULL;
    }

    // Allocate 2D result array for this thread's lines
    long thread_line_count = data->end_line - data->start_line;
    data->thread_results = (char***)malloc(thread_line_count * sizeof(char**));
    if (!data->thread_results) {
        fprintf(stderr, "Thread %d: Failed to allocate thread results array.\n", data->thread_id);
        return NULL;
    }

    for (long i = data->start_line; i < data->end_line; i++) {
        long local_index = i - data->start_line; // Local index within this thread's range
        
        // Initialize context for this line's scan
        MatchContext context;
        context.match_capacity = 16; // Initial capacity
        context.matches = (int*)malloc(context.match_capacity * sizeof(int));
        if (!context.matches) {
             data->thread_results[local_index] = (char**)malloc(sizeof(char*));
             data->thread_results[local_index][0] = strdup(""); // Store empty result on failure
             continue;
        }
        context.match_count = 0;

        // Perform the scan
        hs_error_t err = hs_scan(data->database, data->lines[i], data->line_lengths[i], 0,
                                 data->scratch, onMatch, &context);

        if (err != HS_SUCCESS) {
            free(context.matches);
            data->thread_results[local_index] = (char**)malloc(sizeof(char*));
            data->thread_results[local_index][0] = strdup(""); // Store empty result on error
            continue;
        }

        data->total_matches += context.match_count;

        // Format the result string with ZERO-INDEXED pattern numbers (e.g., "0,3,9")
        if (context.match_count > 0) {
            // A rough estimation for buffer size: 10 chars per match ID + commas
            size_t buffer_size = context.match_count * 10;
            char* result_buffer = (char*)malloc(buffer_size);
            if (!result_buffer) {
                data->thread_results[local_index] = (char**)malloc(sizeof(char*));
                data->thread_results[local_index][0] = strdup("");
            } else {
                int offset = 0;
                for (int j = 0; j < context.match_count; j++) {
                    // Use ZERO-INDEXED pattern numbers (Hyperscan IDs start from 0)
                    offset += snprintf(result_buffer + offset, buffer_size - offset,
                                       "%d%s", context.matches[j], (j == context.match_count - 1) ? "" : ",");
                }
                data->thread_results[local_index] = (char**)malloc(sizeof(char*));
                data->thread_results[local_index][0] = result_buffer;
            }
        } else {
            // If no matches, store an empty string
            data->thread_results[local_index] = (char**)malloc(sizeof(char*));
            data->thread_results[local_index][0] = strdup("");
        }

        free(context.matches);
    }

    // Free scratch space allocated by this thread
    if (data->scratch) {
        hs_free_scratch(data->scratch);
    }

    return NULL;
}


// --- CPU Mode Implementation ---

int run_cpu_mode(const config_t* config) {
    // --- 1. Read and Compile Rules ---
    printf("Reading and compiling regex rules from '%s'...\n", config->rules_file);
    long pattern_count = 0;
    long ignored_total_bytes;
    unsigned int* ignored_lengths;
    char** patterns = read_lines_from_file(config->rules_file, &pattern_count, &ignored_lengths, &ignored_total_bytes);
    free(ignored_lengths);

    unsigned int* ids = (unsigned int*)malloc(pattern_count * sizeof(unsigned int));
    unsigned int* flags = (unsigned int*)malloc(pattern_count * sizeof(unsigned int));
    if (!ids || !flags) fail("Failed to allocate memory for rule IDs/flags.");

    for (long i = 0; i < pattern_count; i++) {
        ids[i] = i; // Hyperscan uses 0-indexed IDs
        flags[i] = 0; // No flags
    }

    hs_database_t* database;
    hs_compile_error_t* compile_err;
    hs_platform_info_t platform;
    
    // Populate platform information for optimal compilation
    hs_error_t platform_err = hs_populate_platform(&platform);
    if (platform_err != HS_SUCCESS) {
        printf("Warning: Could not populate platform info, using default settings.\n");
    }
    
    hs_error_t err = hs_compile_multi((const char* const*)patterns, flags, ids, pattern_count,
                                      HS_MODE_BLOCK, (platform_err == HS_SUCCESS) ? &platform : NULL, 
                                      &database, &compile_err);

    if (err != HS_SUCCESS) {
        fprintf(stderr, "ERROR: Unable to compile pattern: %s\n", compile_err->message);
        hs_free_compile_error(compile_err);
        fail("Hyperscan compilation failed.");
    }
    
    if (!database) {
        fail("Database compilation succeeded but database is NULL.");
    }
    
    printf("Compilation successful. %ld rules loaded.\n", pattern_count);

    // --- 2. Read Input Data ---
    printf("Reading input data from '%s'...\n", config->input_file);
    long line_count = 0;
    long total_bytes = 0;
    unsigned int* line_lengths;
    char** lines = read_lines_from_file(config->input_file, &line_count, &line_lengths, &total_bytes);
    printf("Read %ld lines, total size: %.2f MB.\n", line_count, (double)total_bytes / (1024 * 1024));

    // --- 3. Setup and Run Threads ---
    printf("Processing with %d worker thread(s)...\n", config->num_threads);
    pthread_t* threads = (pthread_t*)malloc(config->num_threads * sizeof(pthread_t));
    ThreadData* thread_data = (ThreadData*)malloc(config->num_threads * sizeof(ThreadData));
    if (!threads || !thread_data) {
        fail("Failed to allocate memory for thread management.");
    }

    struct timespec start_time, end_time;
    clock_gettime(CLOCK_MONOTONIC, &start_time);

    long lines_per_thread = line_count / config->num_threads;
    long remaining_lines = line_count % config->num_threads;
    long current_line = 0;

    for (int i = 0; i < config->num_threads; i++) {
        thread_data[i].thread_id = i;
        thread_data[i].lines = lines;
        thread_data[i].line_lengths = line_lengths;
        thread_data[i].database = database;
        thread_data[i].thread_results = NULL; // Will be allocated by each thread
        thread_data[i].total_matches = 0;
        thread_data[i].scratch = NULL; // Let each thread allocate its own scratch

        // Distribute lines
        thread_data[i].start_line = current_line;
        long chunk_size = lines_per_thread + (i < remaining_lines ? 1 : 0);
        thread_data[i].end_line = current_line + chunk_size;
        current_line += chunk_size;

        pthread_create(&threads[i], NULL, worker_thread, &thread_data[i]);
    }

    // --- 4. Join Threads and Collect Results ---
    long total_matches = 0;
    for (int i = 0; i < config->num_threads; i++) {
        pthread_join(threads[i], NULL);
        total_matches += thread_data[i].total_matches;
    }

    // --- 5. Merge Thread Results into Final Output Array ---
    char** all_results = (char**)malloc(line_count * sizeof(char*));
    if (!all_results) {
        fail("Failed to allocate memory for final results.");
    }

    // Copy results from each thread's 2D array to the final output array
    for (int i = 0; i < config->num_threads; i++) {
        long thread_line_count = thread_data[i].end_line - thread_data[i].start_line;
        for (long j = 0; j < thread_line_count; j++) {
            long global_index = thread_data[i].start_line + j;
            all_results[global_index] = strdup(thread_data[i].thread_results[j][0]);
            
            // Free the thread's result memory
            free(thread_data[i].thread_results[j][0]);
            free(thread_data[i].thread_results[j]);
        }
        free(thread_data[i].thread_results);
    }

    clock_gettime(CLOCK_MONOTONIC, &end_time);
    printf("Processing completed.\n");

    // --- 6. Calculate Performance Metrics ---
    double elapsed_seconds = (end_time.tv_sec - start_time.tv_sec) +
                             (end_time.tv_nsec - start_time.tv_nsec) / 1e9;

    double throughput_input_per_sec = line_count / elapsed_seconds;
    double throughput_mbytes_per_sec = (total_bytes / (1024.0 * 1024.0)) / elapsed_seconds;
    double throughput_match_per_sec = total_matches / elapsed_seconds;
    double latency_ms = (elapsed_seconds * 1000.0) / line_count;

    printf("Performance Metrics:\n");
    printf("  Total Time: %.4f seconds\n", elapsed_seconds);
    printf("  Total Matches: %ld\n", total_matches);
    printf("  Throughput (Input/sec): %.2f\n", throughput_input_per_sec);
    printf("  Throughput (MBytes/sec): %.2f\n", throughput_mbytes_per_sec);
    printf("  Throughput (Match/sec): %.2f\n", throughput_match_per_sec);
    printf("  Latency (ms/input): %.4f\n", latency_ms);

    // --- 7. Write Output Files ---
    char* output_filename = generate_output_filename(config);
    printf("Writing results to '%s'...\n", output_filename);

    // Write match results
    FILE* out_file = fopen(output_filename, "w");
    if (!out_file) fail("Could not open output file for writing.");
    for (long i = 0; i < line_count; i++) {
        fprintf(out_file, "%s\n", all_results[i]);
    }
    fclose(out_file);

    // Write performance metrics
    char* perf_filename = generate_performance_filename(config, config->input_file);
    FILE* perf_file = fopen(perf_filename, "a");
    if (!perf_file) fail("Could not open performance file for writing.");

    // Check if file is empty (new file) to write header
    fseek(perf_file, 0, SEEK_END);
    long file_size = ftell(perf_file);
    if (file_size == 0) {
        // File is empty, write header
        fprintf(perf_file, "threads,throughput_input_per_sec,throughput_mbytes_per_sec,throughput_match_per_sec,latency_ms\n");
    }
    
    fprintf(perf_file, "%d,%.2f,%.2f,%.2f,%.4f\n",
            config->num_threads,
            throughput_input_per_sec,
            throughput_mbytes_per_sec,
            throughput_match_per_sec,
            latency_ms);
    fclose(perf_file);
    
    printf("Results written to '%s' and '%s'\n\n", output_filename, perf_filename);
    free(output_filename);
    free(perf_filename);

    // --- 8. Cleanup ---
    hs_free_database(database);
    for (long i = 0; i < pattern_count; i++) free(patterns[i]);
    free(patterns);
    free(ids);
    free(flags);
    for (long i = 0; i < line_count; i++) {
        free(lines[i]);
        free(all_results[i]);
    }
    free(lines);
    free(line_lengths);
    free(all_results);
    free(threads);
    free(thread_data);

    return EXIT_SUCCESS;
}


// --- GPU Mode Implementation ---

int run_gpu_mode(const config_t* config) {
    printf("Starting GPU mode processing with cuDF/RAPIDS...\n");

    // Memory resources (kept alive for entire scope)
    std::shared_ptr<rmm::mr::cuda_memory_resource> cuda_mr;
    std::shared_ptr<rmm::mr::pool_memory_resource<rmm::mr::cuda_memory_resource>> pool_mr;

    try {
        // --- 1) CUDA & RMM init ---
        CUDA_CHECK(hipSetDevice(0));
        cuda_mr = std::make_shared<rmm::mr::cuda_memory_resource>();
        // 512MB initial pool (grow as needed)
        pool_mr = std::make_shared<rmm::mr::pool_memory_resource<rmm::mr::cuda_memory_resource>>(cuda_mr.get(), 512UL*1024*1024);
        rmm::mr::set_current_device_resource(pool_mr.get());

        hipDeviceProp_t device_prop;
        CUDA_CHECK(hipGetDeviceProperties(&device_prop, 0));
        printf("Using GPU: %s\n", device_prop.name);

        // --- 2) Read patterns ---
        printf("Reading regex patterns from '%s'...\n", config->rules_file);
        long pattern_count = 0;
        unsigned int* tmp_len = nullptr;
        long tmp_bytes = 0;
        char** patterns_c = read_lines_from_file(config->rules_file, &pattern_count, &tmp_len, &tmp_bytes);
        free(tmp_len);
        printf("Loaded %ld patterns.\n", pattern_count);

        std::vector<std::string> patterns;
        patterns.reserve(pattern_count);
        for (long i = 0; i < pattern_count; ++i) patterns.emplace_back(patterns_c[i]);

        // --- 3) Read input lines ---
        printf("Reading input data from '%s'...\n", config->input_file);
        long line_count = 0;
        long total_bytes = 0;
        unsigned int* line_lengths = nullptr;
        char** lines = read_lines_from_file(config->input_file, &line_count, &line_lengths, &total_bytes);
        printf("Read %ld lines, total size: %.2f MB.\n", line_count, (double)total_bytes / (1024.0*1024.0));

        // Global results holder: for each line list of matched pattern IDs
        std::vector<std::vector<int>> line_matches(line_count);

        // --- Timers ---
        struct timespec t0, t1;
        clock_gettime(CLOCK_MONOTONIC, &t0);
        double acc_h2d = 0.0, acc_kernel = 0.0, acc_d2h = 0.0;
        long long total_matches = 0;

        // --- 4) Chunking plan ---
        const long max_mb = (config->gpu_chunk_mb > 0) ? config->gpu_chunk_mb : 128;
        const size_t budget = (size_t)max_mb * 1024ULL * 1024ULL;

        std::vector<std::pair<long,long>> chunks;
        chunks.reserve((size_t)( (total_bytes / (budget?budget:1)) + 2 ));
        long s = 0; size_t acc = 0;
        for (long i = 0; i < line_count; ++i) {
            size_t L = (size_t)line_lengths[i];
            if (i> s && (acc + L) > budget) {
                chunks.emplace_back(s, i);
                s = i; acc = 0;
            }
            acc += L;
        }
        if (s < line_count) chunks.emplace_back(s, line_count);

        printf("Chunking: %zu chunk(s), ~%ld MB each (target).\n", chunks.size(), max_mb);

        // Create one CUDA stream for transfers; compute ops will implicitly use the same stream in this build.
        hipStream_t gpu_stream;
        CUDA_CHECK(hipStreamCreate(&gpu_stream));
        auto stream = rmm::cuda_stream_view{gpu_stream};

        const int wave = (config->gpu_wave > 0) ? config->gpu_wave : 48;

        // --- 5) Process each chunk ---
        for (size_t ci = 0; ci < chunks.size(); ++ci) {
            long lo = chunks[ci].first, hi = chunks[ci].second;
            // Host staging for this chunk
            std::vector<std::string> h_lines;
            h_lines.reserve((size_t)(hi - lo));
            for (long i = lo; i < hi; ++i) h_lines.emplace_back(lines[i]);

            // H2D: build cuDF strings column
            struct timespec h2d_s, h2d_e;
            clock_gettime(CLOCK_MONOTONIC, &h2d_s);
            auto sentences_col = make_device_strings(h_lines, stream);
            cudf::strings_column_view sview{sentences_col->view()};
            int nrows = (int)sview.size();
            CUDA_CHECK(hipStreamSynchronize(stream.value()));
            clock_gettime(CLOCK_MONOTONIC, &h2d_e);
            acc_h2d += (h2d_e.tv_sec - h2d_s.tv_sec) + (h2d_e.tv_nsec - h2d_s.tv_nsec)/1e9;

            // Patterns in waves
            for (long p0 = 0; p0 < pattern_count; p0 += wave) {
                long p1 = std::min(p0 + (long)wave, pattern_count);

                for (long p = p0; p < p1; ++p) {
                    const std::string& pat = patterns[p];
                    if (pat.empty() || pat.size() > 2048) continue;

                    try {
                        // Compile regex program
                        auto prog = cudf::strings::regex_program::create(pat);

                        // Kernel-ish time: contains_re (GPU)
                        struct timespec k_s, k_e;
                        clock_gettime(CLOCK_MONOTONIC, &k_s);
                        auto bool_col = cudf::strings::contains_re(sview, *prog);
                        CUDA_CHECK(hipStreamSynchronize(stream.value()));
                        clock_gettime(CLOCK_MONOTONIC, &k_e);
                        acc_kernel += (k_e.tv_sec - k_s.tv_sec) + (k_e.tv_nsec - k_s.tv_nsec)/1e9;

                        // D2H: bring matches back
                        struct timespec d2h_s, d2h_e;
                        clock_gettime(CLOCK_MONOTONIC, &d2h_s);
                        auto bv = bool_col->view();
                        const uint8_t* d = bv.data<uint8_t>();
                        std::vector<uint8_t> h(bv.size());
                        CUDA_CHECK(hipMemcpyAsync(h.data(), d, h.size(), hipMemcpyDeviceToHost, stream.value()));
                        CUDA_CHECK(hipStreamSynchronize(stream.value()));
                        clock_gettime(CLOCK_MONOTONIC, &d2h_e);
                        acc_d2h += (d2h_e.tv_sec - d2h_s.tv_sec) + (d2h_e.tv_nsec - d2h_s.tv_nsec)/1e9;

                        // Accumulate results
                        long chunk_matches = 0;
                        for (int i = 0; i < nrows; ++i) {
                            if (h[i]) {
                                line_matches[lo + i].push_back((int)p);
                                chunk_matches++;
                            }
                        }
                        total_matches += chunk_matches;
                    } catch (const std::exception& e) {
                        fprintf(stderr, "Warning: pattern %ld failed on chunk %zu: %s\n", p, ci, e.what());
                        CUDA_CHECK(hipStreamSynchronize(stream.value()));
                        continue;
                    }
                } // end for p in wave
            } // end waves per chunk

            // Free the big strings column ASAP
            sentences_col.reset();
            CUDA_CHECK(hipStreamSynchronize(stream.value()));
        } // end chunks

        // --- Assemble all_results like CPU ---
        printf("Formatting results...\n");
        char** all_results = (char**)malloc(line_count * sizeof(char*));
        if (!all_results) fail("Failed to allocate memory for final results.");

        for (long i = 0; i < line_count; ++i) {
            auto& v = line_matches[i];
            if (v.empty()) {
                all_results[i] = strdup("");
            } else {
                // estimate buffer
                size_t buf = v.size()*10;
                char* out = (char*)malloc(buf);
                if (!out) { all_results[i] = strdup(""); continue; }
                int off=0;
                for (size_t j=0;j<v.size();++j) {
                    off += snprintf(out+off, buf-off, "%d%s", v[j], (j+1==v.size())?"":",");
                }
                all_results[i] = out;
            }
        }

        struct timespec t_end; clock_gettime(CLOCK_MONOTONIC, &t_end);
        double elapsed = (t_end.tv_sec - t0.tv_sec) + (t_end.tv_nsec - t0.tv_nsec)/1e9;

        // --- Metrics ---
        double thr_input = line_count / elapsed;
        double thr_mb = (total_bytes / (1024.0*1024.0)) / elapsed;
        double thr_match = (total_matches) / elapsed;
        double latency_ms = (elapsed * 1000.0) / line_count;

        printf("Performance Metrics (GPU):\n");
        printf("  Total Time: %.4f s\n", elapsed);
        printf("  H2D: %.4f s, Kernel: %.4f s, D2H: %.4f s\n", acc_h2d, acc_kernel, acc_d2h);
        printf("  Total Matches: %lld\n", total_matches);
        printf("  Throughput (Input/sec): %.2f\n", thr_input);
        printf("  Throughput (MBytes/sec): %.2f\n", thr_mb);
        printf("  Throughput (Match/sec): %.2f\n", thr_match);
        printf("  Latency (ms/input): %.4f\n", latency_ms);

        // --- Write outputs (same layout) ---
        char* output_filename = generate_output_filename(config);
        printf("Writing results to '%s'...\n", output_filename);
        FILE* out = fopen(output_filename, "w");
        if (!out) fail("Could not open output file for writing.");
        for (long i = 0; i < line_count; ++i) fprintf(out, "%s\\n", all_results[i]);
        fclose(out);

        char* perf_filename = generate_performance_filename(config, config->input_file);
        FILE* pf = fopen(perf_filename, "w");
        if (pf) {
            fprintf(pf, "Mode,DataSet,Library,TotalTime,TotalMatches,InputPerSec,MBPerSec,MatchPerSec,LatencyMs,H2D,Kernel,D2H\\n");
            const char* dataset = strrchr(config->input_file, '/'); dataset = dataset? dataset+1: config->input_file;
            char dataset_clean[256]; strncpy(dataset_clean, dataset, sizeof(dataset_clean)-1); dataset_clean[sizeof(dataset_clean)-1]=0;
            char* dot = strrchr(dataset_clean, '.'); if (dot) *dot = 0;
            fprintf(pf, "GPU,%s,CUDA,%.6f,%lld,%.2f,%.2f,%.2f,%.6f,%.6f,%.6f,%.6f\\n",
                    dataset_clean, elapsed, total_matches, thr_input, thr_mb, thr_match, latency_ms, acc_h2d, acc_kernel, acc_d2h);
            fclose(pf);
        }

        // Cleanup: free C-strings and arrays
        for (long i = 0; i < pattern_count; ++i) free(patterns_c[i]);
        free(patterns_c);
        for (long i = 0; i < line_count; ++i) { free(all_results[i]); }
        free(all_results);
        free(lines);
        free(line_lengths);

        // Destroy stream and RMM pool
        CUDA_CHECK(hipStreamDestroy(gpu_stream));
        rmm::mr::set_current_device_resource(cuda_mr.get());
        pool_mr.reset(); cuda_mr.reset();

        return EXIT_SUCCESS;
    } catch (const std::exception& e) {
        fprintf(stderr, "GPU mode error: %s\\n", e.what());
        if (pool_mr) { rmm::mr::set_current_device_resource(cuda_mr.get()); pool_mr.reset(); }
        if (cuda_mr) cuda_mr.reset();
        return EXIT_FAILURE;
    }
}


// --- Main Function ---

int main(int argc, char* argv[]) {
    config_t config = parse_arguments(argc, argv);
    
    printf("High-Performance Regex Matching - Mode: %s\n", config.mode == MODE_CPU ? "CPU" : "GPU");
    
    if (config.mode == MODE_CPU) {
        return run_cpu_mode(&config);
    } else {
        return run_gpu_mode(&config);
    }
}